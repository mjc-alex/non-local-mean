#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void helloFromGpu() {
	printf("hello from GPU!\n");
}

int main(int argc, char *argv[])
{

	helloFromGpu<<<1,10>>> ();		
	hipDeviceReset();
	return 0;
}
